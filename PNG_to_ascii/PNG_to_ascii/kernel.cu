#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""


#include <iostream>
#include <stdlib.h>
#include <string>

extern "C" {
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
}

using std::string;
using std::cout;
using std::endl;
using std::cin;

const string brightness = "$@B%8&WM#*oahkbdpqwmZO0QLCJUYXzcvunxrjft/\\|()1{}[]?-_ + ~<>i!lI;:,\"^`'.";

__global__ void monochrome(int* width, unsigned char* in, unsigned char* out) {
	const int x = threadIdx.x;
	const int y = threadIdx.y;
	out[y * *width + x] = ((int)in[y * *width + x + 0] + (int)in[y * *width + x + 1] + (int)in[y * *width + x + 2]) / 3;
}

int main(char** argv, int argc) {
	
	string path;
	cin >> path;

	int x, y, n;
	unsigned char* data = stbi_load(path.c_str(), &x, &y, &n, 3);


	unsigned char* cudaData = 0;
	unsigned char* cudaMono = 0;
	int* cudaWidth = 0;


	if (!(data != nullptr && x > 0 && y > 0)) return -1;
	if (n < 3) return -2;

	hipMalloc(&cudaWidth, sizeof(int));
	hipMalloc(&cudaData, x * y * 3);
	hipMalloc(&cudaMono, x * y);

	hipMemcpy(cudaData, data, x * y * 3, hipMemcpyHostToDevice);
	hipMemcpy(cudaWidth, &x, sizeof(int), hipMemcpyHostToDevice);

	unsigned char* mono = new unsigned char[x * y];

	


	int numBlocks = 1;
	dim3 threadsPerBlock(x, y);
	monochrome<<<numBlocks, threadsPerBlock>>>(cudaWidth, cudaData, cudaMono);
	hipMemcpy(mono, cudaMono, x*y, hipMemcpyDeviceToHost);


	for (int i = 0; i < y; i++)
	{
		for (int j = 0; j < x; j++)
		{
			cout << (int)mono[i * x + j] << ", ";
		}
		cout << endl;
	}

	string* output;

	delete mono;
	stbi_image_free(data);

	int height = y / 20;
	int width = x / 20;



}
