#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""


#include <iostream>
#include <stdlib.h>
#include <string>

extern "C" {
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
}

using std::string;
using std::cout;
using std::endl;
using std::cin;

const string brightness = "$@B%8&WM#*oahkbdpqwmZO0QLCJUYXzcvunxrjft/\\|()1{}[]?-_ + ~<>i!lI;:,\"^`'.";

/// <summary>
/// this function turns a pixel in an array of pixels monochrome running on gpu
/// </summary>
/// <param name="width">the horizontal resolution of the image</param>
/// <param name="in">array of rgb pixels, formatted with 3 fields for each pixel.</param>
/// <param name="out">array of monochrome pixels, out param</param>
/// <returns>void</returns>
__global__ void monochrome(const int& width, const unsigned char* in, unsigned char* out) {
	const int x = threadIdx.x;
	const int y = blockIdx.x;
	out[y * width + x] = ((int)in[y * width + x + 0] + (int)in[y * width + x + 1] + (int)in[y * width + x + 2]) / 3;
}


/// <summary>
/// this funtion chunks an array of monochrome pixels into a smaller array of monochrome pixels (downsacling)
/// </summary>
/// <param name="chunkSizeX">the hrizontal amout of pixels to merge</param>
/// <param name="chunkSizeY">the vertical amount of pixel to merge</param>
/// <param name="width">the width of one line of the "in" array</param>
/// <param name="in"> an array filled with monochrome Pixels</param>
/// <param name="out">the downscaled array, out param</param>
/// <returns>void</returns>
__global__ void Chunking(const int& chunkSizeX, const int& chunkSizeY, const int& width, const unsigned char* in, unsigned char* out) {
	const int x = threadIdx.x;
	const int y = blockIdx.x;
	int *sum = new int[chunkSizeY];

	sumChunk<<<1, chunkSizeY>>>(y * width + x, chunkSizeX, width, in, sum);

	int brt = 0;
	for (int i = 0; i < chunkSizeY; i++)
	{
		brt += sum[i];
	}

	brt /= chunkSizeX * chunkSizeY;
	out[x / chunkSizeX, y / chunkSizeY] = brt;
}

/// <summary>
/// this function sums all the values in a given slice of an array
/// </summary>
/// <param name="begins">where the slice begins</param>
/// <param name="length">the length of the slice</param>
/// <param name="width">the horizontal size of the "in" array</param>
/// <param name="in">array of WORD numbers</param>
/// <param name="out">array of WORD numbers, out param</param>
/// <returns>void</returns>
__global__ void sumChunk(const int& begins, const int& length, const int& width, const unsigned char* in, int* out) {
	const int x = threadIdx.x * width + begins;
	int out = 0;
	for (int i = 0; i < length; i++) {
		out[threadIdx.x] += in[x + i];
	}
}

int main(char** argv, int argc) {
	
	string path;
	cin >> path;

	int x, y, n;
	unsigned char* data = stbi_load(path.c_str(), &x, &y, &n, 3);


	unsigned char* cudaData = 0;
	unsigned char* cudaMono = 0;
	int* cudaWidth = 0;


	if (!(data != nullptr && x > 0 && y > 0)) return -1;
	if (n < 3) return -2;

	hipMalloc(&cudaWidth, sizeof(int));
	hipMalloc(&cudaData, x * y * 3);
	hipMalloc(&cudaMono, x * y);

	hipMemcpy(cudaData, data, x * y * 3, hipMemcpyHostToDevice);
	hipMemcpy(cudaWidth, &x, sizeof(int), hipMemcpyHostToDevice);

	unsigned char* mono = new unsigned char[x * y];

	

	dim3 threadsPerBlock(x);
	int numBlocks = y;
	monochrome<<<numBlocks, threadsPerBlock.x>>>(cudaWidth, cudaData, cudaMono);

	hipDeviceSynchronize();

	hipMemcpy(mono, cudaMono, x*y, hipMemcpyDeviceToHost);


	for (int i = 0; i < y; i++)
	{
		for (int j = 0; j < x; j++)
		{
			cout << (int)mono[i * x + j] << ", ";
		}
		cout << endl;
	}

	string* output;

	delete mono;
	stbi_image_free(data);

	int height = y / 20;
	int width = x / 20;



}
