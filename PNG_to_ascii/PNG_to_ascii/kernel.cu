#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""


#include <iostream>
#include <stdlib.h>
#include <string>

extern "C" {
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
}

using std::string;
using std::cout;
using std::endl;

__global__ const string brightness = "$@B%8&WM#*oahkbdpqwmZO0QLCJUYXzcvunxrjft/\\|()1{}[]?-_ + ~<>i!lI;:,\"^`'.";

__global__ void monochrome(int width, int height, int* in, int* out) {
	const int x = threadIdx.x;
	const int y = threadIdx.y;

	out[y * width + x] = (in[y * width + x + 0] + in[y * width + x + 1] + in[y * width + x + 2]) / 3;
}


int main(char** argv, int argc) {
	
	int x, y, n;
	unsigned char* data = stbi_load(argv[1], &x, &y, &n, 3);

	if (!(data != nullptr && x > 0 && y > 0)) return -1;
	if (n != 3) return -2;
	s
	unsigned char* mono = new unsigned char[y * x];

	string* output;

	

	stbi_image_free(data);

	int height = y / 20;
	int width = x / 20;



}
